#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <math.h>
using namespace std;
using namespace std::chrono;

__global__ void AddIntegerArray(int *c, const int *a, const int *b)
{
	int idx = blockIdx.x;
	c[idx] = a[idx] + b[idx];
}

void PrintArray(int *a, int size)
{
	for (int i = 0; i < size; ++i)
		cout << a[i] << ",";
	cout << endl;
}

void CPUAdd(int *c, const int* a, const int* b, int size)
{
	for (int i = 0; i < size; ++i)
		c[i] = a[i] + b[i];
}

void GPUAdd(int *c, const int* a, const int* b, int arraySize)
{
	// allocate memory
	int *d_a, *d_b, *d_c; // = nullptr?
	const int numBytes = arraySize * sizeof(int);
	hipMalloc(&d_a, numBytes);
	hipMalloc(&d_b, numBytes);
	hipMalloc(&d_c, numBytes);

	hipMemcpy(d_a, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, numBytes, hipMemcpyHostToDevice);

	AddIntegerArray << <arraySize, 1 >> > (d_c, d_a, d_b);

	hipMemcpy(c, d_c, numBytes, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

typedef void(adder_t)(int*, const int*, const int*, int);
double TimeFunction(adder_t func, int *c, const int *a, const int *b, int arraySize)
{
	auto start = high_resolution_clock::now();
	func(c, a, b, arraySize);
	auto stop = high_resolution_clock::now();
	return (duration_cast<microseconds>(stop - start)).count();
}

void TestIntegerArrayAddFor(int size)
{
	const int arraySize = size;
	int *a = new int[arraySize];
	int *b = new int[arraySize];
	int *c = new int[arraySize];

	for (int i = 0; i < arraySize; ++i)
	{
		a[i] = i * 10;
		b[i] = i * 15;
	}

	auto cputime = TimeFunction(CPUAdd, c, a, b, arraySize);
	auto gputime = TimeFunction(GPUAdd, c, a, b, arraySize);
	cout << size << "\t|\t" << cputime << "\t|\t" << gputime << endl;
		
	/*PrintArray(a, arraySize);
	PrintArray(b, arraySize);
	PrintArray(c, arraySize);
*/
	delete[] a;
	delete[] b;
	delete[] c;
}

void TestIntegerArrayAdd()
{
	cout << "Size\t|\tCPU\t|\tGPU" << endl;

	TestIntegerArrayAddFor(1); // test
	for (int i = 1; i <= 8; ++i)
		TestIntegerArrayAddFor(pow(10, i));
}
