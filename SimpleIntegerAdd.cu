#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;

__global__ void add(int *c, const int *a, const int *b)
{
	*c = *b + *a;
}

void SimpleIntegerAdd()
{
	cout << "learn cuda cpp" << endl;
	int a, b, c;
	cout << "Enter a & b" << endl;
	cin >> a >> b;

	// create memory
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	// copy to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add << <1, 1 >> >(d_c, d_a, d_b);

	// copy the output
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// free the memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	cout << c;
}